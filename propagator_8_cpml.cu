

#include <hip/hip_runtime.h>
#include <stdio.h>              //manipulacion de ficheros, lectura-escritura ficheros, scandf-printf
#include <stdlib.h>             //Conversion de tipos de datos, memoria dinamica, abs
#include <string.h>             //Uso de memcpy principalmente
#include <math.h>		//funciones matemáticas

#define TILE_WIDTH_X 32
#define TILE_WIDTH_Y 32
#define PI 3.141592653589793
#define left ((ix-1) + Nx*iz)		//Izquierda
#define top  (ix + Nx*(iz-1))		//Arriba
#define center (ix + Nx*iz)		//Centro	
#define bottom (ix + Nx*(iz+1))		//Abajo
#define right  ((ix+1) + Nx*iz)		//Derecha


// A = Presente
// B = Pasado
// C = Futuro

//DEVICE CODE - Kernel1
__global__ void kernel_lap(float *lap, float *A, int nx, int ny, float dh){

	int ix=threadIdx.x + blockIdx.x*blockDim.x;
	int iy=threadIdx.y + blockIdx.y*blockDim.y;

	int tid = ix + iy*nx;

	if(ix > 3 && ix < nx-4 && iy > 3 && iy< ny-4){
		lap[tid] =((-1./560)*(A[ tid - 4 ] + A[tid + 4] + A[tid + 4*nx ] + A[tid -4*nx]) + 
			    (8./315)*(A[ tid - 3 ] + A[tid + 3] + A[tid + 3*nx ] + A[tid -3*nx]) + 
			     (-1./5)*(A[ tid - 2 ] + A[tid + 2] + A[tid + 2*nx ] + A[tid -2*nx]) + 
			      (8./5)*(A[ tid - 1 ] + A[tid + 1] + A[tid + 1*nx ] + A[tid -1*nx]) +
				(-205./36)*(A[tid]))/(dh*dh);
				__syncthreads();
	}

}

__global__ 
void get_CPML_x(float *a_x, float *b_x, int CPMLimit, float R, float VelMax, int Nx, float dt, float dh, float frec){

	int ix = threadIdx.x + blockDim.x * blockIdx.x;    // Indice vector

	float Lx = CPMLimit*dh;
	float d0 = -3*log(R)/(2*Lx);
	a_x[ix] = 0;
	b_x[ix] = 0;
	

	if (ix<CPMLimit+1)	//Left CPML
	{
		b_x[ix] = exp(-( (d0 * VelMax * (((CPMLimit-ix)*dh)/Lx) * (((CPMLimit-ix)*dh)/Lx)) + (PI * frec * (Lx - ((CPMLimit-ix)*dh))/Lx))*dt);
		__syncthreads();
		a_x[ix] = (d0 * VelMax * (((CPMLimit-ix)*dh)/Lx) * (((CPMLimit-ix)*dh)/Lx)) * ( b_x[ix] - 1 ) / ( (d0 * VelMax * (((CPMLimit-ix)*dh)/Lx) * (((CPMLimit-ix)*dh)/Lx)) + (PI * frec * (Lx - ((CPMLimit-ix)*dh))/Lx));
		__syncthreads();
                __syncthreads();		
	}

	if (ix>(Nx-CPMLimit-1) && ix<Nx)	//Right CPML
	{
		b_x[ix] = exp(-( (d0 * VelMax * (((ix-Nx+CPMLimit+1)*dh)/Lx) * (((ix-Nx+CPMLimit+1)*dh)/Lx)) + (PI * frec * (Lx - ((ix-Nx+CPMLimit+1)*dh))/Lx) )*dt);
		__syncthreads();
		a_x[ix] = (d0 * VelMax * (((ix-Nx+CPMLimit+1)*dh)/Lx) * (((ix-Nx+CPMLimit+1)*dh)/Lx)) * ( b_x[ix] - 1 ) / ( (d0 * VelMax * (((ix-Nx+CPMLimit+1)*dh)/Lx) * (((ix-Nx+CPMLimit+1)*dh)/Lx)) + (PI * frec * (Lx - ((ix-Nx+CPMLimit+1)*dh))/Lx) );
		__syncthreads();
                __syncthreads();		
	}

}


__global__ 
void get_CPML_z(float *a_z, float *b_z, int CPMLimit, float R, float VelMax,  int Nz, float dt, float dh, float frec){
	
	//dh = dx; Notacion
	int iz = threadIdx.x + blockDim.x * blockIdx.x;    // Indice vector

	float Lz = CPMLimit*dh;
	float d0 = -3*log(R)/(2*Lz);

	//Inicializando valores de CPML
	a_z[iz] = 0;
	b_z[iz] = 0;

	if (iz>(Nz-CPMLimit-1) && iz<Nz)	//bottom CPML
	{
		b_z[iz] = exp(-( (d0 * VelMax * (((iz-Nz+CPMLimit+1)*dh)/Lz) * (((iz-Nz+CPMLimit+1)*dh)/Lz)) + (PI * frec * (Lz - ((iz-Nz+CPMLimit+1)*dh))/Lz) )*dt);
		__syncthreads();
		a_z[iz] = (d0 * VelMax * (((iz-Nz+CPMLimit+1)*dh)/Lz) * (((iz-Nz+CPMLimit+1)*dh)/Lz)) * ( b_z[iz] - 1 ) / ( (d0 * VelMax * (((iz-Nz+CPMLimit+1)*dh)/Lz) * (((iz-Nz+CPMLimit+1)*dh)/Lz)) + (PI * frec * (Lz - ((iz-Nz+CPMLimit+1)*dh))/Lz) );
		__syncthreads();
	}

}

__global__ 
void PSI(float *A, float *a_x, float *b_x, float *a_z, float *b_z, float *Psi_x, float *Psi_z, int CPMLimit, int Nx, int Nz, float dh){

	int ix = threadIdx.x + blockDim.x * blockIdx.x;		// Row  of  the  A matrix
	int iz = threadIdx.y + blockDim.y * blockIdx.y;		// Column of the A matrix
	int tid = ix + iz*Nx;


	if(ix > 3 && ix < Nx-4 && iz > 3 && iz< Nz-4)
	{
    // Primera derivada de segundo orden centrada
  /*    
		Psi_x[tid] = Psi_x[tid]*b_x[ix] + a_x[ix]*( (-1./2)*A[tid-1] + (1./2)*A[tid+1])/(2*dh);
		Psi_z[tid] = Psi_z[tid]*b_z[iz] + a_z[iz]*( (-1./2)*A[tid-Nx]  + (1./2)*A[tid+Nx])/(2*dh);
  */  
    // Primera derivada de octavo orden centrada
		
	Psi_x[tid] = Psi_x[tid]*b_x[ix] + a_x[ix]*( (1./280.)*A[tid-4] - (4./105.)*A[tid-3] + (1./5.)*A[tid-2] - (4./5.)*A[tid-1] + (4./5.)*A[tid+1] + (-1./5.)*A[tid+2] + (4./105.)*A[tid+3] + (-1./280.)*A[tid+4] )/(dh);
		__syncthreads();	  

	Psi_z[tid] = Psi_z[tid]*b_z[iz] + a_z[iz]*( (1./280.)*A[tid-4*Nx] + (-4./105.)*A[tid-3*Nx] + (1./5.)*A[tid-2*Nx] + (-4./5.)*A[tid-1*Nx] + (4./5.)*A[tid+1*Nx] + (-1./5.)*A[tid+2*Nx] + (4./105.)*A[tid+3*Nx] + (-1./280.)*A[tid+4*Nx] )/(dh);
		__syncthreads();	  
  }
}


__global__ 
void kernel_propaga(float *lap, float *A, float *B, int nx, int nz, float *source, float *c, float *traza, float *P, float *dP, int it, int sx, int sz, int borde, float dt, float *a_z, float *a_x, float *b_z, float *b_x, float *Psi_x, float *Psi_z, float *Z_x, float *Z_z, float *temp, float dh, int modo){

	int ix=threadIdx.x + blockIdx.x*blockDim.x;
	int iy=threadIdx.y + blockIdx.y*blockDim.y;
	int tid  = ix + iy*nx;
	float G  = c[tid]*c[tid]*dt*dt;

  
	if(ix > 3 && ix < nx-4 && iy > 3 && iy< nz-4){
    // Calculo de las derivadas de psi

     // Primera derivada de segundo orden central

     //temp[tid]=((-1./2)*Psi_x[tid-1] + (1./2)*Psi_x[tid+1])/(2*dh) + ((-1./2)*Psi_z[tid-nx] + (1./2)*Psi_z[tid+nx])/(2*dh); 
     // Primera derivada octavo orden central
    temp[tid]=( (1./280)*Psi_x[tid-4] + (-4./105)*Psi_x[tid-3] + (1./5)*Psi_x[tid-2] + (-4./5)*Psi_x[tid-1] 
	     + (4./5)*Psi_x[tid+1] + (-1./5)*Psi_x[tid+2] + (4./105)*Psi_x[tid+3] + (-1./280)*Psi_x[tid+4] )/(dh) 
	     + ( (1./280)*Psi_z[tid-4*nx] +  (-4./105)*Psi_z[tid-3*nx] + (1./5)*Psi_z[tid-2*nx] + (-4./5)*Psi_z[tid-1*nx] 
	     + (4./5)*Psi_z[tid+1*nx] + (-1./5)*Psi_z[tid+2*nx] + (4./105)*Psi_z[tid+3*nx] + (-1./280)*Psi_z[tid+4*nx] )/(dh); 
    __syncthreads();
    
// Calculo de los zetas

    // Derivadas de segundo orden
    // Z_x[tid]=b_x[ix]*Z_x[tid] + a_x[ix]*( (A[tid+1] -2*A[tid] + A[tid-1])/(dh*dh) + ((-1./2)*Psi_x[tid-1] + (1./2)*Psi_x[tid+1])/dh);
    // Z_z[tid]=b_z[iy]*Z_z[tid] + a_z[iy]*( (A[tid-nx] -2*A[tid] + A[tid+nx])/(dh*dh) +  ((-1./2)*Psi_z[tid-nx] + (1./2)*Psi_z[tid+nx])/dh);

    // Derivadas de octavo orden
    Z_x[tid]=b_x[ix]*Z_x[tid] + a_x[ix]*( ( (-1./560)*A[tid-4] + (8./315)*A[tid-3] + (-1./5)*A[tid-2] + (8./5)*A[tid-1] + (-205./72)*A[tid] + (8./5)*A[tid+1] + (-1./5)*A[tid+2] + (8./315)*A[tid+3] + (-1./560)*A[tid+4] )/(dh*dh) + ((1./280)*Psi_x[tid-4] + (-4./105)*Psi_x[tid-3] + (1./5)*Psi_x[tid-2] + (-4./5)*Psi_x[tid-1] + (4./5)*Psi_x[tid+1] + (-1./5)*Psi_x[tid+2] + (4./105)*Psi_x[tid+3] + (-1./280)*Psi_x[tid+4] )/(dh*dh));
    __syncthreads();
    
    Z_z[tid]=b_z[iy]*Z_z[tid] + a_z[iy]*( ( (-1./560)*A[tid-4*nx] + (8./315)*A[tid-3*nx] + (-1./5)*A[tid-2*nx] + (8./5)*A[tid-1*nx] + (-205./72)*A[tid] + (8./5)*A[tid+1*nx] + (-1./5)*A[tid+2*nx] + (8./315)*A[tid+3*nx] + (-1./560)*A[tid+4*nx] )/(dh*dh) + ((1./280)*Psi_z[tid-4*nx] + (-4./105)*Psi_z[tid-3*nx] + (1./5)*Psi_z[tid-2*nx] + (-4./5)*Psi_z[tid-1*nx] + (4./5)*Psi_z[tid+1*nx] + (-1./5)*Psi_z[tid+2*nx] + (4./105)*Psi_z[tid+3*nx] + (-1./280)*Psi_z[tid+4*nx] )/(dh*dh));
    __syncthreads();
    
  }

	//if(ix<nx-1 && iy<nz-1){
	if(ix > 3 && ix < nx-4 && iy > 3 && iy< nz-4 ){
 		B[tid]=2*A[tid] - B[tid] + G*( lap[tid] + Z_x[tid] + Z_z[tid] + temp[tid]);
		__syncthreads();
	}

	if(ix < nx && iy< nz){
//		if(it<Nt){
			if(ix==(sx-1+4) && iy==(sz-1+4)){
				A[tid] = A[tid] + source[it];
				__syncthreads();
			}
//		}

	if(modo==1 || modo==3 ){
		P[tid+nx*nz*it]=B[tid];
	  __syncthreads();
	}
	if(modo==2 || modo==3 ){
		dP[tid+nx*nz*it]=c[tid]*c[tid]*(lap[tid] + Z_x[tid] + Z_z[tid] + temp[tid]);
	  __syncthreads();
	}
	if(ix>=borde && ix< nx-borde && iy==sz-1+4){
		traza[(ix-borde)+it*(nx-2*borde)]=B[tid];
	__syncthreads();
	}
	}
	__syncthreads();
}
// A = Presente
// B = Pasado
// C = Futuro
//HOST CODE
int main(){
 hipDeviceReset();
  //variables host
	
	int borde=20;
  	int nx=210;
	int ny=71;
	int modo=1;
	int it;
	int nt;
	int sx = ceil(nx/2);
	int sz = 6;
  	float VelMax=4700;
	float R=100e-6;
	float frec=3;
	float tend=1.0;
	float dt=0.001; 
	float dh=25;
	float  *A_d, *B_d, *Pt_d, *Pt_h, *P_d, *P_h, *dP_d, *dP_h, *s_h, *s_d, *v_d, *v_h, *A_x, *A_z, *B_x, *B_z, *lap, *temp, *temp1, *Psi_x, *Psi_z, *Z_x, *Z_z;
	FILE *source, *model_ori;

	nt= ceil(tend/dt);

	Pt_h = (float *)calloc((nx-2*borde)*nt,sizeof(float));
	v_h  = (float *)calloc(nx*ny,sizeof(float));
	s_h  = (float *)calloc(nt,sizeof(float));
	P_h  = (float *)calloc(nx*ny*nt,sizeof(float));
	dP_h = (float *)calloc(nx*ny*nt,sizeof(float));

  //variables y memory allocation en device
	hipMalloc(&Pt_d,(nx-2*borde)*nt*sizeof(float));
        hipMalloc(&A_d, nx*ny*sizeof(float));
        hipMalloc(&B_d, nx*ny*sizeof(float));
        hipMalloc(&P_d, nx*ny*nt*sizeof(float));
        hipMalloc(&dP_d, nx*ny*nt*sizeof(float));
        hipMalloc(&s_d, nt*sizeof(float));
        hipMalloc(&v_d, nx*ny*sizeof(float));
        hipMalloc(&lap, nx*ny*sizeof(float));
        hipMalloc(&Psi_x, nx*ny*sizeof(float));
        hipMalloc(&Psi_z, nx*ny*sizeof(float));
        hipMalloc(&Z_x, nx*ny*sizeof(float));
        hipMalloc(&Z_z, nx*ny*sizeof(float));
        hipMalloc(&temp1, nx*ny*sizeof(float));

        hipMalloc(&A_x, nx*sizeof(float));
        hipMalloc(&A_z, ny*sizeof(float));
        hipMalloc(&B_x, nx*sizeof(float));
        hipMalloc(&B_z, ny*sizeof(float));
      	
        hipMemset(A_d,0,nx*ny*sizeof(float));
      	hipMemset(B_d,0,nx*ny*sizeof(float));
      	hipMemset(lap,0,nx*ny*sizeof(float));
        hipMemset(Psi_x,0,nx*ny*sizeof(float));
        hipMemset(Psi_z,0,nx*ny*sizeof(float));
        hipMemset(Z_x,0,nx*ny*sizeof(float));
        hipMemset(Z_z,0,nx*ny*sizeof(float));
        hipMemset(temp1,0,nx*ny*sizeof(float));

        hipMemset(A_x,0,nx*sizeof(float));
        hipMemset(A_z,0,ny*sizeof(float));
        hipMemset(B_x,0,nx*sizeof(float));
        hipMemset(B_z,0,ny*sizeof(float));


	//Leer y condicionar archivos fuente 
        source = fopen ("Fuente.bin","rb");
        fread(s_h,nt*sizeof(float),1,source);
        printf("\nDatos de fuente cargados...\n");
        fclose(source);

        //Leer y condicionar archivos modelo original
        model_ori = fopen ("Modelo_ori.bin","rb");
        fread(v_h,nx*ny*sizeof(float),1,model_ori);
        printf("\nDatos de modelo original cargados...\n");
        fclose(model_ori);
	
	//Enviando informacion necesaria a la GPU
        hipMemcpy(v_d, v_h, nx*ny*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(s_d, s_h, nt*sizeof(float), hipMemcpyHostToDevice);
	

        //ejecución Kernel
        dim3 dimGrid_M(ceil((nx) / (float)TILE_WIDTH_X) , ceil((ny) / (float)TILE_WIDTH_Y));
        dim3 dimBlock_M(TILE_WIDTH_X , TILE_WIDTH_Y);

      	dim3 dimGrid_Vx(ceil((nx) / (float)TILE_WIDTH_X));
        dim3 dimBlock_Vx(TILE_WIDTH_X);

      	dim3 dimGrid_Vz(ceil((ny) / (float)TILE_WIDTH_Y));
        dim3 dimBlock_Vz(TILE_WIDTH_Y);

        get_CPML_x<<<dimGrid_Vx, dimBlock_Vx>>>(A_x, B_x, borde, R, VelMax, nx, dt, dh, frec);
        get_CPML_z<<<dimGrid_Vz, dimBlock_Vz>>>(A_z, B_z, borde, R, VelMax, ny, dt, dh, frec);

	for (it=0;it<nt;it++){
		printf("Voy en el paso temporal %d \n",it+1);
          
          PSI<<<dimGrid_M, dimBlock_M>>>(A_d, A_x, B_x, A_z, B_z, Psi_x, Psi_z, borde, nx, ny, dh);
          kernel_lap<<<dimGrid_M, dimBlock_M>>>(lap,A_d,nx,ny,dh); //#blocks=dimGrid, #threads=dimBlock
          kernel_propaga<<<dimGrid_M, dimBlock_M>>>(lap,A_d,B_d,nx,ny,s_d,v_d,Pt_d,P_d,dP_d,it,sx,sz,borde,dt,A_z,A_x,B_z,B_x,Psi_x,Psi_z,Z_x,Z_z,temp1,dh,modo); //#blocks=dimGrid,
          temp = A_d;
          A_d = B_d;
          B_d = temp;
	}
	
	if(modo==1 || modo==3){
	
        	hipMemcpy(P_h, P_d, nx*ny*nt*sizeof(float), hipMemcpyDeviceToHost);
	}
	
	if(modo==2 || modo==3){

        	hipMemcpy(dP_h, dP_d, nx*ny*nt*sizeof(float), hipMemcpyDeviceToHost);
	}

        hipMemcpy(Pt_h, Pt_d, (nx-2*borde)*nt*sizeof(float), hipMemcpyDeviceToHost);
        //---------------------------------------
        // Guarda informacion -----------------------
	// Trazas
	source=fopen("Trazas.bin","wb");
	fwrite(Pt_h,sizeof(float),(nx-2*borde)*nt,source);
	fclose(source);
	free(Pt_h);
	hipFree(Pt_d);
	// Frente de Onda
	source=fopen("Frentedeonda.bin","wb");
	fwrite(P_h,sizeof(float),nx*ny*nt,source);
	fclose(source);
	free(P_h);
	hipFree(P_d);

	// Derivad Frente de Onda 
	source=fopen("DerivadaFrentedeOnda.bin","wb");
	fwrite(dP_h,sizeof(float),nx*ny*nt,source);
	fclose(source);
	free(dP_h);
	hipFree(dP_d);

	// Liberamos resto de punteros
	// Host
  	free(s_h);
	free(v_h);
	// Device
	hipFree(A_d);
	hipFree(B_d);
	hipFree(lap);
	hipFree(s_d);
	hipFree(v_d);
	hipFree(A_x);
	hipFree(A_z);
	hipFree(B_x);
	hipFree(B_z);
	hipFree(Psi_x);
	hipFree(Psi_z);
	hipFree(Z_x);
	hipFree(Z_z);

	printf("\n Creo que Termine ....\n");

 hipDeviceReset();
  return 0;
}
